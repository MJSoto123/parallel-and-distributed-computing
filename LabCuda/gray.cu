#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>

#define BLUR_SIZE 3
#define CHANNELS 3

using namespace cv;

__global__
void colorToGreyscaleConversion(unsigned char* out, unsigned char* in, int w, int h) 
{	
	int Col = threadIdx.x + blockIdx.x * blockDim.x;
	int Row = threadIdx.y + blockIdx.y * blockDim.y;
	if (Col < w && Row < h) 
    {
		int greyOffset = Row * w + Col;
		
		int offset = greyOffset * CHANNELS;
		unsigned char smr = in[offset];  
		unsigned char smg = in[offset + 1]; 
		unsigned char smb = in[offset + 2]; 

		out[offset] = 0.21f * smr + 0.71f * smg + 0.07f * smb;
		out[offset+1] = 0.21f * smr + 0.71f * smg + 0.07f * smb;
		out[offset+2] = 0.21f * smr + 0.71f * smg + 0.07f * smb;
	}
}

int main(int argc, char** argv) {
    if (argc != 2) {
        std::cerr << "Uso: " << argv[0] << " <nombre_de_archivo_imagen>" << std::endl;
        return -1;
    }

    int w, h;
    Mat image1 = imread(argv[1], IMREAD_COLOR);

    if (image1.empty()) {
        std::cerr << "Error al cargar la imagen." << std::endl;
        return -1;
    }

    namedWindow("Imagen Original", WINDOW_AUTOSIZE);
    imshow("Imagen Original", image1);
    waitKey(0);

    Size imageSize = image1.size();
    w = imageSize.width;
    h = imageSize.height;

    std::cout<<w<<" "<<h<<"\n";

    unsigned char* ptrImageData = NULL;
    unsigned char* ptrImageDataOut = NULL;

    hipMalloc(&ptrImageDataOut, w * h * CHANNELS);
    hipMalloc(&ptrImageData, w * h * CHANNELS);
    hipMemcpy(ptrImageData, image1.data, w * h * CHANNELS, hipMemcpyHostToDevice);

    dim3 dimGrid(ceil(w / 16.0), ceil(h / 16.0), 1);
    dim3 dimBlock(16, 16, 1);

    colorToGreyscaleConversion<<<dimGrid, dimBlock>>>(ptrImageDataOut, ptrImageData, w, h);
    hipDeviceSynchronize(); // Esperar a que todos los bloques terminen

    Mat image2(h, w, CV_8UC3);
    hipMemcpy(image2.data, ptrImageDataOut, w * h * CHANNELS, hipMemcpyDeviceToHost);

    std::string nuevoNombre = argv[1];
    nuevoNombre = nuevoNombre.substr(0, nuevoNombre.find_last_of('.')) + "_toGray.png";
    imwrite(nuevoNombre, image2);

    hipFree(ptrImageData);
    hipFree(ptrImageDataOut);

    namedWindow("Imagen Procesada", WINDOW_AUTOSIZE);
    imshow("Imagen Procesada", image2);
    waitKey(0);

    return 0;
}
