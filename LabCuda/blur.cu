#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>

#define BLUR_SIZE 3
#define CHANNELS 3

using namespace cv;

__global__
void colorToBlurConversion(uc* out, uc* in, int w, int h) 
{
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    int Row = blockIdx.y * blockDim.y + threadIdx.y;

    if (Col < w && Row < h) {
		int smr = 0;
		int smg = 0;
		int smb = 0;
		int pixels=0;
		int Offset = (Row * w + Col) * CHANNELS;

		for (int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE + 1; ++blurRow) 
        {
			for (int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE + 1; ++blurCol) 
            {
				int curRow = Row + blurRow;
				int curCol = Col + blurCol;
				if (curRow > -1 && curRow < h && curCol > -1 && curCol < w) 
                {
					int current = (curRow * w + curCol) * CHANNELS;
					smr += in[current];
					smg += in[current + 1];
					smb += in[current + 2];
					pixels++;
				}
			}
		}
		out[Offset] = (unsigned char)(smr / pixels);
		out[Offset + 1] = (unsigned char)(smg / pixels);
		out[Offset + 2] = (unsigned char)(smb / pixels);
	}
}

int main(int argc, char** argv) {
    if (argc != 2) {
        std::cerr << "Uso: " << argv[0] << " <nombre_de_archivo_imagen>" << std::endl;
        return -1;
    }

    int w, h;
    Mat image1 = imread(argv[1], IMREAD_COLOR);

    if (image1.empty()) {
        std::cerr << "Error al cargar la imagen." << std::endl;
        return -1;
    }

    namedWindow("Imagen Original", WINDOW_AUTOSIZE);
    imshow("Imagen Original", image1);
    waitKey(0);

    Size imageSize = image1.size();
    w = imageSize.width;
    h = imageSize.height;

    std::cout<<w<<" "<<h<<"\n";

    uc* ptrImageData = NULL;
    uc* ptrImageDataOut = NULL;

    hipMalloc(&ptrImageDataOut, w * h * CHANNELS);
    hipMalloc(&ptrImageData, w * h * CHANNELS);
    hipMemcpy(ptrImageData, image1.data, w * h * CHANNELS, hipMemcpyHostToDevice);

    dim3 dimGrid(ceil(w / 16.0), ceil(h / 16.0), 1);
    dim3 dimBlock(16, 16, 1);

    colorToBlurConversion<<<dimGrid, dimBlock>>>(ptrImageDataOut, ptrImageData, w, h);
    hipDeviceSynchronize(); // Esperar a que todos los bloques terminen

    Mat image2(h, w, CV_8UC3);
    hipMemcpy(image2.data, ptrImageDataOut, w * h * CHANNELS, hipMemcpyDeviceToHost);

    std::string nuevoNombre = argv[1];
    nuevoNombre = nuevoNombre.substr(0, nuevoNombre.find_last_of('.')) + "_toBlur.jpeg";
    imwrite(nuevoNombre, image2);

    hipFree(ptrImageData);
    hipFree(ptrImageDataOut);

    namedWindow("Imagen Procesada", WINDOW_AUTOSIZE);
    imshow("Imagen Procesada", image2);
    waitKey(0);

    return 0;
}